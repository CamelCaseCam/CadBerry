#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <stdbool.h> 
#include "frna.h"
#include "../partition-smp/util.h"

//#define DISABLE_COAXIAL
//struct frna
//{
//  int n; /* number of bases */
//  fbase_t *seq;  /* sequence */
//  int_t *v; /* n x n array */
//  int_t *w, *wm, *wca; /* n x n arrays */
//  int_t *w5, *w3; /* n elements */
//};

/* penalty for a helix terminated by a pair containing a U */
DEV static int_t terminal_U_penalty(const fbase_t *s, const int i, const int j, fparam_t p)
{
  return s[i] == U || s[j] == U ? p->terminal_AU_penalty : 0.0;
}

DEV static int_t dangle_3p_energy(const fbase_t *s,
				   const int i,
				   const int j,
				   const int ip1,
                                   fparam_t p)
{
  return p->dangle_3p[s[i]][s[j]][s[ip1]] + terminal_U_penalty(s,i,j,p);
}

DEV static int_t dangle_5p_energy(const fbase_t *s,
				   const int i,
				   const int j,
				   const int jm1,
                                   fparam_t p)
{
  return p->dangle_5p[s[i]][s[j]][s[jm1]] + terminal_U_penalty(s,i,j,p);
}

DEV static int_t terminal_stack(const fbase_t *s,
		                 const int i,
				 const int j,
				 const int ip1,
				 const int jm1,
                                 fparam_t p)
{
  return p->tstack[s[i]][s[j]][s[ip1]][s[jm1]] + terminal_U_penalty(s,i,j,p);
}

DEV static int_t terminal_stack_multibranch(const fbase_t *s,
					     const int i,
					     const int j,
					     const int ip1,
					     const int jm1,
					     fparam_t p)
{
  return p->tstackm[s[i]][s[j]][s[ip1]][s[jm1]] + terminal_U_penalty(s,i,j,p);
}


DEV static const int_t *lookup_find(const fbase_t *s, const int d, fparam_t p)
{
  int i;
  switch (d) {
  case 3:
    for (i = 0; i < p->ntriloop; i++)
      if (sequences_match(s, p->triloop[i].seq, d+2))
	return &p->triloop[i].val;
    break;
  case 4:
    for (i = 0; i < p->ntloop; i++)
      if (sequences_match(s, p->tloop[i].seq, d+2))
	return &p->tloop[i].val;
    break;
  case 6:
    for (i = 0; i < p->nhexaloop; i++)
      if (sequences_match(s, p->hexaloop[i].seq, d+2))
	return &p->hexaloop[i].val;
    break;
  }
  return 0;
}

/***
 * Energy of a hairpin loop with d unpaired bases, d = j-i-1
 * s[i] is paired with s[j]
 * s[i+1] is mismatched with s[j-1]
 ***/
DEV static int_t hairpin_loop_energy(const fbase_t *s,
				      const int i,
				      const int j,
				      const int d,
                                      fparam_t p)
{
  /* Lookup tables for special hairpin loops */
  const int_t *val;
  if ((val = lookup_find(&s[i],d,p)))
    return *val;

  /* Hairpin loop initiation penalty */
  int_t e;
  if (d > LOOP_MAX)
    e = (int_t) (p->hairpin_loop_initiation[LOOP_MAX] + p->prelog *
      LOG((float) d / LOOP_MAX));
  else
    e = p->hairpin_loop_initiation[d];

  if (d == 3) {
    if (contains_only_base(C,d,&s[i+1]))
      e += p->c_hairpin_of_3;
    e += terminal_U_penalty(s,i,j,p);
  } else {
    e += p->tstackh[s[i]][s[j]][s[i+1]][s[j-1]];
    if (contains_only_base(C,d,&s[i+1]))
      e += p->c_hairpin_slope*d + p->c_hairpin_intercept;
  }

  if (s[i] == G && s[j] == U && i > 1 && s[i-1] == G && s[i-2] == G)
    e += p->bonus_for_GGG_hairpin;

  return e;

}

DEV static int_t real_min(int_t a, int_t b) { return a < b ? a : b; }

/***
 * Energy of an internal/bulge loop with d1, d2 unpaired bases,
 *   d1 = ip-i-1,  d2 = j-jp-1
 * s[i] is paired with s[j]
 * s[i+1] is mismatched sith s[j-1]
 * s[ip-1] is mismatched with s[jp+1]
 * s[ip] is paired with s[jp]
 ***/

DEV static int_t alternative_bulge_loop_correction (const int n, const fbase_t *s,
                                                  const int i,
                                                  const int ip) //i<ip
{
  int count = 1;
  int k;
  //float result;
  if (i!=n-1){
    k = i;
    while (k>=0 && s[k]==s[i+1]) {
      count++;
      k--;
    }

    k = ip;
    while (k<=n-1 && (s[k]==s[i+1])) {
      count++;
      k++;
    }
  }
  return (int_t) (-1.0f * RT * conversion_factor * log ((float) count));
}

DEV static int_t internal_loop_energy(const fbase_t *s,
                                       const int n,
				       const int i,
				       const int j,
				       const int ip,
				       const int jp,
				       const int d1,
				       const int d2,
                                       fparam_t p)
{
  /* Bulge loops */
  if (d1 == 0 || d2 == 0) {
    int_t e = p->bulge_loop_initiation[d1+d2];
    if (d1 == 1 || d2 == 1) { /* single-nucleotide bulge */
      e += p->stack[s[i]][s[j]][s[ip]][s[jp]];
        if (d1==0) e += alternative_bulge_loop_correction(n,s,jp,j); //correction for multiple equivalent bulge loops
        //else e += alternative_bulge_loop_correction(s,i,jp);
        else e += alternative_bulge_loop_correction(n,s,i,ip);
      if ((d1 == 1 && s[i+1] == C && (s[i] == C || s[i+2] == C)) ||
          (d2 == 1 && s[j-1] == C && (s[j] == C || s[j-2] == C)))
        e += p->Bonus_for_Single_C_bulges_adjacent_to_C;
    } else {
      e += terminal_U_penalty(s,i,j,p);
      e += terminal_U_penalty(s,ip,jp,p);
    }
    return e;
  }

  /* Small internal loops */
  if (d1 == 1 && d2 == 1)
    return p->int11[s[i]][s[i+1]][s[i+2]][s[j-2]][s[j-1]][s[j]];
  if (d1 == 2 && d2 == 2)
    return p->int22[s[i]][s[ip]][s[j]][s[jp]][s[i+1]][s[i+2]][s[j-1]][s[j-2]];
  if (d1 == 1 && d2 == 2)
    return p->int21[s[i]][s[j]][s[i+1]][s[j-1]][s[jp+1]][s[ip]][s[jp]];
  if (d1 == 2 && d2 == 1)
    return p->int21[s[jp]][s[ip]][s[jp+1]][s[ip-1]][s[i+1]][s[j]][s[i]];

  /* Larger internal loops */
  tab4_t *sp;
  if (d1 == 1 || d2 == 1)
    sp = &p->tstacki1n;
  else if ((d1 == 2 && d2 == 3) || (d1 == 3 && d2 == 2))
    sp = &p->tstacki23;
  else
    sp = &p->tstacki;
  return p->internal_loop_initiation[d1+d2] +
    real_min(p->fm_array_first_element*abs(d1-d2), p->maximum_correction) +
    (*sp)[s[i]][s[j]][s[i+1]][s[j-1]] +
    (*sp)[s[jp]][s[ip]][s[jp+1]][s[ip-1]];

}

#ifndef disablecoax
DEV static int_t coaxial_flush(const fbase_t *s,
				const int i,
				const int j,
				const int ip,
				const int jp,
                                fparam_t p)
{
  return terminal_U_penalty(s,i,j,p) + terminal_U_penalty(s,ip,jp,p) +
    p->coaxial[s[i]][s[j]][s[ip]][s[jp]];
}

DEV static int_t coaxial_mismatch1(const fbase_t *s,
				    const int i,
				    const int j,
				    const int ip,
				    const int jp,
                                    fparam_t p)
{
  return terminal_U_penalty(s,i,j,p) + terminal_U_penalty(s,ip,jp,p) +
    p->tstackcoax[s[j]][s[i]][s[j+1]][s[i-1]] +
    p->coaxstack[s[j+1]][s[i-1]][s[ip]][s[jp]];
}

DEV static int_t coaxial_mismatch2(const fbase_t *s,
				    const int i,
				    const int j,
				    const int ip,
				    const int jp,
                                    fparam_t p)
{
  return terminal_U_penalty(s,i,j,p) + terminal_U_penalty(s,ip,jp,p) +
    p->tstackcoax[s[jp]][s[ip]][s[jp+1]][s[ip-1]] +
    p->coaxstack[s[j]][s[i]][s[j+1]][s[jp+1]];
}
#endif//disablecoax

DEV static void free_energy_min(int_t *a, const int_t b)
{
 if(*a>b) *a = b;
}

DEV HOST static int int_min(int a, int b) { return a < b ? a : b; }

DEV HOST static int_t int_t_min(int_t a, int_t b) { return a < b ? a : b; }

DEV HOST static int ind(int i, int j, int n)
{
  return i*n + j;
}

DEV HOST inline static int cp(int i, int j, const fbase_t *s)
{
  return j-i-1 >= LOOP_MIN && is_canonical_pair(s[i],s[j]);
}

DEV HOST inline static int can_pair(int i, int j, int n, const fbase_t *s)
{
  if (j < i) {
    const int tmp = i;
    i = j;
    j = tmp;
  }
  return cp(i,j,s) && ((i > 0 && j < n-1 && cp(i-1,j+1,s)) || cp(i+1,j-1,s));
}

DEV HOST inline static int not_isolated(int i,int j,int n, const fbase_t *s)
{
  if (j < i) {
    const int tmp = i;
    i = j;
    j = tmp;
  }
  return is_canonical_pair(s[i],s[j]) && ((i > 0 && j < n-1 && cp(i-1,j+1,s)) || cp(i+1,j-1,s));
}

DEV static int wrap(int i, int n)
{
  return i >= n ? i-n : i;
}

DEV static int is_exterior(int i, int j)
{
  return j < i;
}

DEV static int is_interior(int i, int j)
{
  return i < j;
}

DEV HOST static int_t *array_val(int_t *a, int i, int j, int n, const fbase_t *s)
{
  return can_pair(i,j,n,s) ? &a[ind(i,j,n)] : 0;
//  return &a[ind(i,j,n)];
}

#ifdef __HIPCC__
#define ISTART blockIdx.x
#define IINC gridDim.x
#else
#define ISTART 0
#define IINC 1
#endif

//MFE recursions begin
//TODO
//figure out source of differences in arrays
//integrate with rnastructure traceback


//when recursions work on the cpu:
//do the same thing with the calculation on the GPU
GLOBAL static void calc_V_hairpin_and_V_stack//_multibranch//calculate V(i,j)=Vhairpin+Vstack+Vmultibranch but NOT bulge/internal
(int d, int n, fbase_t *s, int_t *v, int_t *w, int_t *wm, int_t *w5, int_t *w3, fparam_t p)
{
  int i;
  for (i = ISTART; i < n; i += IINC) { //for(i=blockId.x;i<numberofbases;i+=gridDim.x) so each thread will handle 0+blockId.x,256+blockId.x..
    const int jtmp = i+d+1;
    const int j = wrap(jtmp,n);
    if((is_interior(i,j) && !can_pair(i,j,n,s)) || (is_exterior(i,j) && (!is_canonical_pair(s[i],s[j]) ))){
      v[ind(i,j,n)] = INF; //this is important
      continue;
    }
    int_t vij = INF; //temp variable to fold free energy sum
    if (i != n-1 && j != 0) {
      /* hairpin loop */
      if (is_interior(i,j))
        vij = hairpin_loop_energy(s,i,j,d,p);
      /* stack */
      //if (can_pair(i+1,j-1,n,s) && !((is_interior(i,j)) && (d <= LOOP_MIN-2)))//-2???
      if (can_pair(i+1,j-1,n,s) && !((is_interior(i,j)) && (d <= LOOP_MIN-2)))//-2???
        free_energy_min(&vij, p->stack[s[i]][s[j]][s[i+1]][s[j-1]] + v[ind(i+1,j-1,n)]);
    }
    v[ind(i,j,n)] = vij;
  }
}



#ifdef __HIPCC__

#define NTHREAD 256
#define SQRT_NTHREAD 16

DEV static void free_energy_min_reduce(int_t *x, int tid, int nt)
{
  __shared__ int_t buf[NTHREAD];
  buf[tid] = *x;
  for (nt /= 2, __syncthreads(); nt > 0; nt /= 2, __syncthreads())
    if (tid < nt)
      free_energy_min(&buf[tid], buf[tid+nt]);
  if (tid == 0)
    *x = buf[0];
}

#endif /* __HIPCC__ */
GLOBAL static void calc_V_bulge_internal (int d, int n, fbase_t *s, int_t *v, int_t *w, int_t *wl, int_t *w5, int_t *w3, fparam_t p)
{
//	Vbi(i,j) = min[V(k,l)+ Ebulge/int(i,j,k,l)] where i<k<l<j, i!=i+1, and j!=j-1
  int i;
  for (i = ISTART; i < n; i += IINC) {
    const int jtmp = i+d+1;
    const int j = wrap(jtmp,n);
    if ((is_exterior(i,j) && i-j <= LOOP_MIN) ||
        (is_interior(i,j) && d <= LOOP_MIN+2) ||
        !can_pair(i,j,n,s))
      continue;
    int_t vij = INF;
#ifdef __HIPCC__
    const int d1start = threadIdx.x;
    const int d1inc = blockDim.x;
#else
    const int d1start = 0;
    const int d1inc = 1;
#endif
    const int dmax = int_min(LOOP_MAX, d-2);
    const int d1max = int_min(dmax, n-i-2);
    int d1;
    for (d1 = d1start; d1 <= d1max; d1 += d1inc) { //d1start is threadid, d1max is max loop size
      const int ip = i+d1+1; //ip depends on thread's ID in x dimension
      const int d2max = int_min(dmax-d1, j-1);
#ifdef __HIPCC__
      const int d2start = d1 > 0 ? threadIdx.y : threadIdx.y + 1;
      const int d2inc = blockDim.y;
#else
      const int d2start = d1 > 0 ? 0 : 1;
      const int d2inc = 1;
#endif
      int d2;
      for (d2 = d2start; d2 <= d2max; d2 += d2inc) {
        const int jp = j-d2-1;//jp depends on thread's ID in the y dimension
        if (can_pair(ip,jp,n,s))
          free_energy_min(&vij, internal_loop_energy(s,n,i,j,ip,jp,d1,d2,p) + v[ind(ip,jp,n)]);
      }
    }
#ifdef __HIPCC__
    const int tid = threadIdx.x * blockDim.y + threadIdx.y;
    free_energy_min_reduce(&vij, tid, blockDim.x*blockDim.y); //after we have 1 value per thread, do parallel reduction
    if (tid != 0)
      continue;
#endif
    free_energy_min(&v[ind(i,j,n)], vij); //write vij to V
  }

}

GLOBAL static void calc_V_multibranch (int d, int n, fbase_t *s, int_t *v, int_t *w, int_t *wm, int_t *w5, int_t *w3, fparam_t p)
{

//	Vmb(i,j) = min[WM(i+1,j-1)+c+a, WM(i+2,j-1)+Edangle5'+a+b+c, WM(i+1,j-2)+Edangle3'+a+b+c, WM(i+2,j-2)+Edangleboth+a+2b+c,
//		min_over_k[ V(i+1,k) + min[W(k+1,j-1), WM(k+1,j-1)]] + a+2c+Eflushcoax(i to j, i+1 to k) , //various coaxial stacking possibilities
//		min_over_k[ V(k,j-1) + min[W(i+1,k-1), WM(i+1,k-1)]] + a+2c+Eflushcoax(i to j, k to j-1) ,
//		min_over_k[ V(i+2,k) + min[W(k+2,j-1), WM(k+2,j-1)]] + a+2c+2b+Emismatch3'coax(i to j, i+2 to k) ,
//		min_over_k[ V(i+2,k) + min[W(k+1,j-2), WM(k+1,j-2)]] + a+2c+2b+Emismatch5'coax(i to j, i+2 to k) ,
//		min_over_k[ V(k,j-2) + min[W(i+2,k-1), WM(i+2,k-1)]] + a+2c+2b+Emismatch3'coax(i to j, k to j-2) ,
//		min_over_k[ V(k,j-2) + min[W(i+1,k-2), WM(i+1,k-2)]] + a+2c+2b+Emismatch5'coax(i to j, k to j-2) ]

//	where i < k < j


	//V(i,j) = min(V(i,j), Vmb(i,j))

  int i;
  for (i = ISTART; i < n; i += IINC) { //for(i=blockId.x;i<numberofbases;i+=gridDim.x) so each thread will handle 0+blockId.x,256+blockId.x..
    const int jtmp = i+d+1;
    const int j = wrap(jtmp,n);
    if ((is_exterior(i,j) && i-j <= LOOP_MIN) || !can_pair(i,j,n,s))
      continue;
    int_t vij=INF;
    if (d > 2*LOOP_MIN + 3 && i != n-1 && j != 0) { //if i and j are far enough apart to close a MBL..
        free_energy_min(&vij, wm[ind(i+1,j-1,n)] + terminal_U_penalty(s,i,j,p) + p->a + p->c);
        if (i != n-2)
          free_energy_min(&vij, wm[ind(i+2,j-1,n)] + dangle_3p_energy(s,i,j,i+1,p) + p->a + p->b + p->c);
        if (j != 1)
          free_energy_min(&vij, wm[ind(i+1,j-2,n)] + dangle_5p_energy(s,i,j,j-1,p) + p->a + p->b + p->c);
        if (i != n-2 && j != 1)
          free_energy_min(&vij, wm[ind(i+2,j-2,n)] + terminal_stack_multibranch(s,i,j,i+1,j-1,p) + p->a + 2*p->b + p->c);
        /*
         * this is done in calc_coaxial
#ifndef disablecoax
        //6 coaxial stacking possibilities
        int k;
        int_t vcoax = INF;
        for(k=i+1;k<j;k++){
          const int a2c = p->a + 2*p->c;
          free_energy_min(&vcoax,v[ind(i+1,k,n)] + wmin(k+1,j-1,n,w,wm) + coaxial_flush(s,i,j,i+1,k,p) + a2c);
          free_energy_min(&vcoax,v[ind(k,j-1,n)] + wmin(i+1,k-1,n,w,wm) + coaxial_flush(s,i,j,k,j-1,p) + a2c);
          free_energy_min(&vcoax,v[ind(i+2,k,n)] + wmin(k+2,j-1,n,w,wm) + coaxial_mismatch1(s,i,j,i+2,k,p) + a2c);
          free_energy_min(&vcoax,v[ind(i+2,k,n)] + wmin(k+1,j-2,n,w,wm) + coaxial_mismatch2(s,i,j,i+2,k,p) + a2c);
          free_energy_min(&vcoax,v[ind(k,j-2,n)] + wmin(i+2,k-1,n,w,wm) + coaxial_mismatch1(s,i,j,k,j-1,p) + a2c);
          free_energy_min(&vcoax,v[ind(k,j-2,n)] + wmin(i+1,k-2,n,w,wm) + coaxial_mismatch2(s,i,j,k,j-1,p) + a2c);
        }
        free_energy_min(&vij,vcoax);
#endif
*/
    }
  free_energy_min(&v[ind(i,j,n)], vij);
  }
}


GLOBAL static void calc_V_exterior (int d, int n, fbase_t *s, int_t *v, int_t *w, int_t *wl, int_t *w5, int_t *w3, fparam_t p)
{
//	Vexterior(i,j) = min[ W3(i+1)+W3(j-1-N), W3(i+2)+W5(j-1-N)+E5'dangle, W3(i+1)+W5(j-2-N)+E3'dangle, W3(i+2)+W5(j-2-N)+Emismatch,
//		min_over_k[ V(i+1,k) + W3(k+1) + W5(j-1-N) + Eflushcoax ],
//		min_over_k[ V(k,j-1-N) + W3(i+1) + W5(k-1) + E ],
//		min_over_k[ V(i+2,k-2) + W3(k+1) + W5(j-1-N) + E ],
//		min_over_k[ V(i+2,k-1) + W3(k+1) + W5(j-2-N) + E ],
//		min_over_k[ V(k+1,j-2-N) + W3(i+1) + W5(k-1) + E ],
//		min_over_k[ V(k,j-2-N) + W3(i+2) + W5(k-1) + E ] ]

  int i;
  for (i = ISTART; i < n; i += IINC) { //for(i=blockId.x;i<numberofbases;i+=gridDim.x) so each thread will handle 0+blockId.x,256+blockId.x..
    const int jtmp = i+d+1;
    const int j = wrap(jtmp,n);
//    if ( !is_interior(i,j) || (i-j <= LOOP_MIN) || !can_pair(i,j,n,s))
    if ( is_interior(i,j))
    //if ( is_interior(i,j) )//wrong answer
      continue;
    int_t vij = INF; //temp variable to fold free energy sum
    if(is_canonical_pair(s[i],s[j])&&not_isolated(i,j,n,s)){
      free_energy_min(&vij, w3[i+1] + w5[j-1] + terminal_U_penalty(s,i,j,p));
      if (i != n-1)
        free_energy_min(&vij, w3[i+2] + w5[j-1] + dangle_3p_energy(s,i,j,i+1,p));
      if (j != 0)
        free_energy_min(&vij, w3[i+1] + w5[j-2] + dangle_5p_energy(s,i,j,j-1,p));
      if (i != n-1 && j != 0)
        free_energy_min(&vij, w3[i+2] + w5[j-2] + terminal_stack(s,i,j,i+1,j-1,p));
    }
    /*
     * this is done in calc_coaxial
#ifndef disablecoax
    int k;
    //for k from 0 to j-1
    for(k=j-1;k>0;k--){
    //not at all sure about subscripts. algorithm line 3815
      free_energy_min(&vij,w3[i+1]+w5[k-1]+coaxial_flush(s,k,j-1,j,i,p));
      //RNAstructure has condition j-number-2>0
      free_energy_min(&vij,w3[i+2]+w5[k-1]+coaxial_mismatch2(s,k,j-2,j,i,p));
      free_energy_min(&vij,w3[i+1]+w5[k-1]+coaxial_mismatch1(s,k+1,j-2,j,i,p));
    }
    //for k from i+1 to n
    for(k=i+1;k<n;k++){
      //algorithm:3875
      //needs conditions
      free_energy_min(&vij,w3[k+1]+w5[j-1]+coaxial_flush(s,j,i,i+1,k,p));
      free_energy_min(&vij,w3[k+1]+w5[j-2]+coaxial_mismatch1(s,j,i,i+2,k,p));
      free_energy_min(&vij,w3[k+1]+w5[j-1]+coaxial_mismatch2(s,j,i,i+2,k-1,p));
    }
#endif //disablecoax
*/
    free_energy_min(&v[ind(i,j,n)], vij);
    }
}

GLOBAL static void calc_W (int d, int n, fbase_t *s, int_t *v, int_t *w, int_t *wl, int_t *w5, int_t *w3, fparam_t p)
{
	//W(i,j) = min[V(i,j)+c,V(i+1,j)+Edangle5',
//			V(i,j+1)+Edangle3',
//			V(i+1,j+1)+Edangleboth]

  int i;
  for (i = ISTART; i < n; i += IINC) { //for(i=blockId.x;i<numberofbases;i+=gridDim.x) so each thread will handle 0+blockId.x,256+blockId.x..
    const int jtmp = i+d+1;
    const int j = wrap(jtmp,n);
    int_t wij = INF; //temp variable to fold free energy sum
    int_t* v_temp;
    //consider adding nucleotide to existing loop
    if(d>0){
      if (i!=n-1)
        free_energy_min(&wij, w[ind(i+1,j,n)] + p->b);
      if(j!=0)
        free_energy_min(&wij, w[ind(i,j-1,n)] + p->b);
    }
    if((is_interior(i,j) && (d>LOOP_MIN-1)) ){
      v_temp = array_val(v,i,j,n,s);
        free_energy_min(&wij, (v_temp? *v_temp:INF) + terminal_U_penalty(s,i,j,p) + p->c);
      if(j!=0){
        v_temp = array_val(v,i,j-1,n,s);
        free_energy_min(&wij, (v_temp? *v_temp:INF) + dangle_3p_energy(s,j-1,i,j,p) + p->b + p->c);
      }

      if(i!=n-1) {
        v_temp = array_val(v,i+1,j,n,s);
        free_energy_min(&wij, (v_temp? *v_temp:INF) + dangle_5p_energy(s,j,i+1,i,p) + p->b + p->c);
        }

      if((i!=n-1) && (j!=0)){
        v_temp = array_val(v,i+1,j-1,n,s);
        free_energy_min(&wij, (v_temp? *v_temp:INF) + terminal_stack_multibranch(s,j-1,i+1,j,i,p) + 2*p->b + p->c);
      }
    }
    if(is_exterior(i,j)){
      free_energy_min(&wij, v[ind(i,j,n)] + terminal_U_penalty(s,i,j,p) + p->c);
      if(j!=0){
        //v_temp = array_val(v,i,j-1,n,s);
        free_energy_min(&wij, v[ind(i,j-1,n)] + dangle_3p_energy(s,j-1,i,j,p) + p->b + p->c);
      }

      if(i!=n-1) {
        //v_temp = array_val(v,i+1,j,n,s);
        free_energy_min(&wij, v[ind(i+1,j,n)] + dangle_5p_energy(s,j,i+1,i,p) + p->b + p->c);
        }

      if((i!=n-1) && (j!=0)){
        //v_temp = array_val(v,i+1,j-1,n,s);
        free_energy_min(&wij, v[ind(i+1,j-1,n)] + terminal_stack_multibranch(s,j-1,i+1,j,i,p) + 2*p->b + p->c);
      }
    }
    w[ind(i,j,n)] = wij;
  }
}


GLOBAL static void calc_WM (int d, int n, fbase_t *s, int_t *v, int_t *w, int_t *wm, int_t *w5, int_t *w3, fparam_t p)
{
	//WM(i,j) = min[W(i,k)+W(k+1,j),
//			V(i,k)+V(k+1,j)+2c+Eflushcoax,
//			V(i,k)+V(k+2,j-1)+2c+Ecoax5'mismatch,
//			V(i+1,k)+V(k+2,j)+2c+Ecoax3'mismatch]

  int i;
  for (i = ISTART; i < n; i += IINC) { //for(i=blockId.x;i<numberofbases;i+=gridDim.x) so each thread will handle 0+blockId.x,256+blockId.x..
    const int jtmp = i+d+1;
    const int j = wrap(jtmp,n);
    int_t tmp = INF;

//don't need to calculate every WM
  if((is_interior(i,j) && (j-i-1 <= 2*LOOP_MIN+2))){//condition copied verbatim from algorithm.cpp
    wm[ind(i,j,n)]=INF;
    continue;
  }
//  if(is_interior(i,j)) printf("i %d j %d\n",i,j);


#ifdef __HIPCC__
    const int kstart = i + threadIdx.x;
//    const int kstart = i+1 + threadIdx.x;
    const int kinc = blockDim.x;
#else
    const int kstart = i;
//    const int kstart = i+1;
    const int kinc = 1;
#endif
    int ktmp;
//    for (ktmp = kstart; ktmp < jtmp-1; ktmp += kinc) {
    for (ktmp = kstart; ktmp < jtmp; ktmp += kinc) {
      if (ktmp != n-1) {
        const int k = wrap(ktmp,n);
        free_energy_min(&tmp, w[ind(i,k,n)] + w[ind(k+1,j,n)]);
#ifndef disablecoax
    /*
        //TODO add condition that i and k can pair
        const int c2 = 2*p->c;
        free_energy_min(&tmp, v[ind(i,k,n)] + v[ind(k+1,j,n)] + coaxial_flush(s,i,k,k+1,j,p) + c2);
        //i-1 in mismatch
        free_energy_min(&tmp, v[ind(i,k,n)] + v[ind(k+2,j,n)] + coaxial_mismatch1(s,i,k,k+2,j,p) + c2);
        ///j+1 in mismatch
        free_energy_min(&tmp, v[ind(i,k,n)] + v[ind(k+2,j,n)] + coaxial_mismatch2(s,i,k,k+2,j,p) + c2);
        */
#endif//disablecoax
      }

    }

    if(d>0){
      if (i!=n-1)
        free_energy_min(&tmp, wm[ind(i+1,j,n)] + p->b);
      if(j!=0)
        free_energy_min(&tmp, wm[ind(i,j-1,n)] + p->b);
    }

#ifdef __HIPCC__
    free_energy_min_reduce(&tmp, threadIdx.x, blockDim.x);
    if (threadIdx.x != 0)
      continue;
#endif
    wm[ind(i,j,n)] = tmp;
    free_energy_min(&w[ind(i,j,n)],tmp);
  }
}

#ifndef disablecoax
GLOBAL static void calc_coaxial (int d, int n, fbase_t *s, int_t *v, int_t *w, int_t *wm, int_t *w5, int_t *w3, fparam_t p)
{
  int i;
  for (i = ISTART; i < n; i += IINC) {
    const int jtmp = i+d+1;
    const int j = wrap(jtmp,n);
    if ((is_exterior(i,j) && i-j <= LOOP_MIN) || !can_pair(i,j,n,s))
      continue;
    const int_t *v1;
    int_t vij = INF;
    /* exterior */
    if (is_exterior(i,j)) {
      int k, kstart;
#ifdef __HIPCC__
      kstart = threadIdx.x;
      const int kinc = blockDim.x;
#else
      kstart = 0;
      const int kinc = 1;
#endif
      for (k = kstart; k < j - LOOP_MIN; k += kinc) {
	if ((v1 = array_val(v,k,j-1,n,s)))
	  free_energy_min(&vij, w3[i+1] + w5[k-1] + coaxial_flush(s,k,j-1,j,i,p) + (*v1));
	if (j-2 >= 0) {
	  if (i < n-1 && (v1 = array_val(v,k,j-2,n,s)))
	    free_energy_min(&vij, w3[i+2] + w5[k-1] + coaxial_mismatch2(s,k,j-2,j,i,p) + (*v1));
	  if ((v1 = array_val(v,k+1,j-2,n,s)))
	    free_energy_min(&vij, w3[i+1] + w5[k-1] + coaxial_mismatch1(s,k+1,j-2,j,i,p) + (*v1));
	}
      }
#ifdef __HIPCC__
      kstart = i+LOOP_MIN+1 + threadIdx.x;
#else
      kstart = i+LOOP_MIN+1;
#endif
      for (k = kstart; k < n; k += kinc) {
	if ((v1 = array_val(v,i+1,k,n,s)))
	  free_energy_min(&vij, w3[k+1] + w5[j-1] + coaxial_flush(s,j,i,i+1,k,p) + (*v1));
	if (j > 0 && (v1 = array_val(v,i+2,k,n,s)))
	  free_energy_min(&vij, w3[k+1] + w5[j-2] + coaxial_mismatch1(s,j,i,i+2,k,p) + (*v1));
	if ((v1 = array_val(v,i+2,k-1,n,s)))
	  free_energy_min(&vij, w3[k+1] + w5[j-1] + coaxial_mismatch2(s,j,i,i+2,k-1,p) + (*v1));
      }
    } /* end exterior */

    /* multibranch */
    if (d > 2*LOOP_MIN + 3 && i != n-1 && j != 0) {
      int ktmp;
#ifdef __HIPCC__
      int ktmpstart = i+2 + threadIdx.x;
      const int ktmpinc = blockDim.x;
#else
      int ktmpstart = i+2;
      const int ktmpinc = 1;
#endif
      for (ktmp = ktmpstart; ktmp < jtmp-2; ktmp += ktmpinc) {
	const int k = wrap(ktmp,n);
	if (k != n-1) {
	  if ((v1 = array_val(v,i+1,k,n,s)))
	    free_energy_min(&vij, coaxial_flush(s,j,i,i+1,k,p) + (*v1) + p->a_2c +
				   w[ind(k+1,j-1,n)]);
	  if (ktmp+2 < jtmp-1 && i+1 != n-1 && k+1 != n-1 && (v1 = array_val(v,i+2,k,n,s))) {
	    const int_t tmp = (*v1) + p->a_2b_2c;
	    free_energy_min(&vij, coaxial_mismatch2(s,j,i,i+2,k,p) + tmp + w[ind(k+2,j-1,n)]);
	    if (j != 1) {
	      free_energy_min(&vij, coaxial_mismatch1(s,j,i,i+2,k,p) + tmp + w[ind(k+1,j-2,n)]);
	    }
	  }
	}
      }
#ifdef __HIPCC__
      ktmpstart = i+3 + threadIdx.x;
#else
      ktmpstart = i+3;
#endif
      for (ktmp = ktmpstart; ktmp < jtmp-1; ktmp += ktmpinc) {
	const int k = wrap(ktmp,n);
	if (k != 0) {
	  if ((v1 = array_val(v,k,j-1,n,s)))
	    free_energy_min(&vij, coaxial_flush(s,k,j-1,j,i,p) + (*v1) + p->a_2c +
				   w[ind(i+1,k-1,n)]);
	  if (j != 1 && ktmp > i+3 && (v1 = array_val(v,k,j-2,n,s))) {
	    const int_t tmp = (*v1) + p->a_2b_2c;
	    if (k != 1)
	      free_energy_min(&vij, coaxial_mismatch1(s,k,j-2,j,i,p) + tmp + w[ind(i+1,k-2,n)]);
	    if (i != n-2)
	      free_energy_min(&vij, coaxial_mismatch2(s,k,j-2,j,i,p) + tmp + w[ind(i+2,k-1,n)]);
	  }
	}
      }
    } /* end multibranch */
#ifdef __HIPCC__
    free_energy_min_reduce(&vij, threadIdx.x, blockDim.x);
    if (threadIdx.x != 0)
      continue;
#endif
    free_energy_min(&v[ind(i,j,n)], vij);
  } /* end loop over i */
} /* end calc_coaxial */

GLOBAL static void calc_wl_coax(int d, int n, fbase_t *s, int_t *v, int_t *w, int_t *wm, int_t *w5, int_t *w3, fparam_t p, int_t *wca)
{
  int i;
  for (i = ISTART; i < n; i += IINC) {
    const int jtmp = i+d+1;
    const int j = wrap(jtmp,n);
    if ((is_exterior(i,j) && i-j <= LOOP_MIN) ||
	(is_interior(i,j) && d <= 2*LOOP_MIN+1))
      continue;
#ifdef __HIPCC__
    const int kstart = i+LOOP_MIN+1 + threadIdx.x;
    const int kinc = blockDim.x;
#else
    const int kstart = i+LOOP_MIN+1;
    const int kinc = 1;
#endif
    int ktmp;
    int_t tmp1 = INF, tmp2 = INF;
    for (ktmp = kstart; ktmp < jtmp-LOOP_MIN-1; ktmp += kinc) {
      const int k = wrap(ktmp,n);
      if (k == n-1) continue;
      int_t *v1, *v2;
      if ((v1 = array_val(v,i,k,n,s)) && (v2 = array_val(v,k+1,j,n,s))){
        free_energy_min(&tmp1, (*v1) + (*v2) + coaxial_flush(s,i,k,k+1,j,p));
      }
      if (j == 0 || k+1 == n-1) continue;
      if (i != n-1 && (v1 = array_val(v,i+1,k,n,s)) && (v2 = array_val(v,k+2,j,n,s))){
        free_energy_min(&tmp2, (*v1) + (*v2) + coaxial_mismatch1(s,i+1,k,k+2,j,p));
      }
      if ((v1 = array_val(v,i,k,n,s)) && (v2 = array_val(v,k+2,j-1,n,s))){
        free_energy_min(&tmp2, (*v1) + (*v2) + coaxial_mismatch2(s,i,k,k+2,j-1,p));
      }
    }
#ifdef __HIPCC__
    free_energy_min_reduce(&tmp1, threadIdx.x, blockDim.x);
    free_energy_min_reduce(&tmp2, threadIdx.x, blockDim.x);
    if (threadIdx.x != 0) continue;
#endif
//    if (is_interior(i,j))
//      free_energy_accumulate(&wq[upper_triangle_index(i,j)], free_energy_sum(tmp1,tmp2));
//    tmp1+=2*p->c;
//    tmp2+=(2*p->b+2*p->c);
//    const int_t wcoax = (tmp1 + 2*p->c, tmp2 + 2*p->b + 2*p->c);
//    const int_t wcoax=tmp1>tmp2? tmp2:tmp1;
    wca[ind(i,j,n)] = int_t_min(tmp1,tmp2);
    free_energy_min(&wm[ind(i,j,n)], tmp1+2*p->c);
    free_energy_min(&wm[ind(i,j,n)], tmp2+2*p->b+2*p->c);
    free_energy_min(&w[ind(i,j,n)], wm[ind(i,j,n)]);
  } /* end loop over i */
} /* end calc_z */

#endif /* disablecoax */


GLOBAL static void calc_w5_and_w3 (int d, int n, fbase_t *s, int_t *v, int_t *w, int_t *wm, int_t *w5, int_t *w3, fparam_t p, int_t* wca)
{
#ifdef __HIPCC__
  const int istart = threadIdx.x;
  const int iinc = blockDim.x;
#else
  const int istart = 0;
  const int iinc = 1;
#endif
//  int_t w5tmp = INF, w3tmp = INF;//this should probably be initialized to zero
  int_t w5tmp=0,w3tmp = 0;
  int i;
  int_t* v_temp;
  for (i = istart; i + LOOP_MIN <= d; i += iinc) {

    if((v_temp = array_val(v,i,d+1,n,s)))
//      free_energy_min(&w5tmp, w5[i-1] + *v_temp + terminal_U_penalty(s,i,d+1,p));
      free_energy_min(&w5tmp, w5[i-1] + *v_temp + terminal_U_penalty(s,d+1,i,p)); //the nucleotide thats more 3' has to go first in terminal_U_penalty call
    if(d-i>LOOP_MIN){//necessary, or we seg fault because we try to have a pair in a 4mer
//      if((d!=n-2) && (v_temp = array_val(v,i,d,n,s))) //d!=n-2 condition actually keeps you from considering a 3' dangle on the end of the sequence
      if((v_temp = array_val(v,i,d,n,s)))
        //free_energy_min(&w5tmp, w5[i-1] + *v_temp + dangle_3p_energy(s,i,d+1,d,p));
        free_energy_min(&w5tmp, w5[i-1] + *v_temp + dangle_3p_energy(s,d,i,d+1,p));
      if((v_temp = array_val(v,i+1,d+1,n,s)))
        //free_energy_min(&w5tmp, w5[i-1] + *v_temp + dangle_5p_energy(s,i+1,d+1,i,p));
        free_energy_min(&w5tmp, w5[i-1] + *v_temp + dangle_5p_energy(s,d+1,i+1,i,p));
#ifndef disablecoax
      free_energy_min(&w5tmp,w5[i-1] + wca[ind(i,d+1,n)]);
#endif
    }
//    if ((d-i>LOOP_MIN+1) && (d!=n-2) && (v_temp = array_val(v,i+1,d,n,s)))//playing with conditions here
    if ((d-i>LOOP_MIN+1) && ((v_temp = array_val(v,i+1,d,n,s))))
      free_energy_min(&w5tmp, w5[i-1] + *v_temp + terminal_stack(s,d,i+1,d+1,i,p));
    //  free_energy_min(&w5tmp, w5[i-1] + *v_temp + terminal_stack(s,i,d+1,i+1,d,p));//wrong arg order

    if((v_temp = array_val(v,n-d-2,n-i-1,n,s)))
      free_energy_min(&w3tmp, w3[n-i] + *v_temp + terminal_U_penalty(s,n-i-1,n-d-2,p));
    if((v_temp = array_val(v,n-d-2,n-i-2,n,s)))
      free_energy_min(&w3tmp, w3[n-i] + *v_temp + dangle_3p_energy(s,n-i-2,n-d-2,n-i-1,p));
    if((n-d-1 != 0) && ((v_temp = array_val(v,n-d-1,n-i-1,n,s))))
      free_energy_min(&w3tmp, w3[n-i] + *v_temp + dangle_5p_energy(s,n-i-1,n-d-1,n-d-2,p));
    if((n-i-2 != n-1) && (n-d-1 != 0) && ((v_temp = array_val(v,n-d-1,n-i-2,n,s))))
      free_energy_min(&w3tmp, w3[n-i] + *v_temp + terminal_stack(s,n-i-2,n-d-1,n-i-1,n-d-2,p));
#ifndef disablecoax
    free_energy_min(&w3tmp,w3[n-i] + wca[ind(n-d-2,n-i-1,n)]);
#endif
  }
#ifdef __HIPCC__
  free_energy_min_reduce(&w5tmp, threadIdx.x, blockDim.x);
  free_energy_min_reduce(&w3tmp, threadIdx.x, blockDim.x);
  if (threadIdx.x != 0)
    return;
#endif
//}
  w5[d+1] = w5[d];
  w3[n-d-2] = w3[n-d-1];
  free_energy_min(&w5[d+1], w5tmp);
  free_energy_min(&w3[n-d-2], w3tmp);
} /* end calc_w5_and_w3 */


GLOBAL static void init_w5_and_w3 (int n,int_t *w5, int_t *w3)
{
#ifdef __HIPCC__
  w5[blockIdx.x] = 0;
  w3[blockIdx.x] = 0;
#else
  int i;
  for(i=0;i<n+1;i++){
    w5[i] = 0;
    w3[i] = 0;
  }
#endif
//  w5[-1]=0;
//  w5[0]=w3[n-1]=w3[n]=0;
}

GLOBAL static void init_arr(int n,int_t *arr)
{
  int i = threadIdx.x+blockIdx.x*blockDim.x;
  arr[i] = INF;
//  int j;
//  for(i=0;i<n;i++)
//	for(j=0;j<n;j++)
//	    w[i]=INF;
}

//MFE recursions end

void initialize(int_t* arr,size_t size){
    size_t i;
    for(i=0;i<size;i++){
        arr[i] = INF;
    }
}

frna_t frna_new(const char *s, fparam_t par)
{
  frna_t p = (frna_t) safe_malloc(sizeof(struct frna));
  memset(p, 0, sizeof(struct frna));

  const int n = p->n = strlen(s);
  p->seq = fsequence_from_string(s);
  p->v = (int_t *) safe_malloc(n*n*sizeof(int_t));
  p->w = (int_t *) safe_malloc(n*n*sizeof(int_t));
  p->wm = (int_t *) safe_malloc(n*n*sizeof(int_t));
  p->wca = (int_t *) safe_malloc(n*n*sizeof(int_t));
  p->w5 = (int_t *) safe_malloc((n+1)*sizeof(int_t)) + 1;
//  p->w5[-1]=0;
  p->w3 = (int_t *) safe_malloc((n+1)*sizeof(int_t));
  initialize(p->v,n*n);
  initialize(p->w,n*n);
  initialize(p->wm,n*n);
  initialize(p->wca,n*n);
  //initialize(p->w5,n+1);
  //initialize(p->w3,n+1);



#ifdef __HIPCC__ /* do multithreaded fill on GPU */

  int_t *v,*w,*wm,*w5,*w3,*wca;
  hipEvent_t start, stop, tstart, tstop;
  float t;
  CU(hipEventCreate(&start));
  CU(hipEventCreate(&stop));
  CU(hipEventCreate(&tstart));
  CU(hipEventCreate(&tstop));

  CU(hipEventRecord(start,0));

//  int_t *v, *w5, *w3;

#define ALLOC(a,sz) CU(hipMalloc(&a,(sz)*sizeof(int_t)))

  ALLOC(v,n*n); //best energy of structure closed by pair i,j. j>i: exterior fragment
  ALLOC(w,n*n); //best energy of structure from i to j
  ALLOC(wm,n*n); //best energy of structure i to j containing 2 or more branches
  ALLOC(w5,n+1); //best energy of structure from 1 to i
  w5++;//w5 is indexed from 1 -- is this a good idea?
  ALLOC(w3,n+1); //best energy of structure from i to numberofbases
  ALLOC(wca,n*n);

/*
  ALLOC(z,n*n);
  ALLOC(yl,n*n);
  ALLOC(y,n*n);
  ALLOC(wq,n*(n-1)/2);
  ALLOC(w,2*n);
  ALLOC(wl,2*n);
  ALLOC(xl,2*n);
  ALLOC(x,5*n);*/

  fparam_t dev_par;
  CU(hipMalloc(&dev_par, sizeof(struct fparam)));
  CU(hipMemcpy(dev_par, par, sizeof(struct fparam), hipMemcpyHostToDevice));

  fbase_t *dev_s;
  CU(hipMalloc(&dev_s,n*sizeof(fbase_t)));
  CU(hipMemcpy(dev_s, p->seq, n*sizeof(fbase_t), hipMemcpyHostToDevice));

  CU(hipEventRecord(stop,0));
  CU(hipEventSynchronize(stop));
  CU(hipEventElapsedTime(&t,start,stop));
#ifdef TIMER
  fprintf(stderr, "Time for copy from CPU to GPU: %3.1f ms\n", t);
#endif

  int d;
  float t1=0, t2=0, t3=0, t4=0, t5=0, t6=0, t7=0, t8=0, tcoax=0, tcoax2=0;

//HACK HACK HACK
//THIS MEMORY REALLY NEEDS TO BE INITIALIZED WITH A REASONABLE INT_T VALUE
  hipMemset(v,50,n*n*sizeof(int_t));
  hipMemset(w,50,n*n*sizeof(int_t));
  hipMemset(wm,50,n*n*sizeof(int_t));
  hipMemset(wca,50,n*n*sizeof(int_t));
  hipMemset(w5-1,0,(n+1)*sizeof(int_t));
  hipMemset(w3,0,(n+1)*sizeof(int_t));
  init_w5_and_w3<<<n,1>>>(n+1,w5-1,w3);
/*
  init_arr<<<n,n>>>(n*n,v);
  init_arr<<<n,n>>>(n*n,wm);
  init_arr<<<n,n>>>(n*n,w);
  init_arr<<<n,n>>>(n*n,wca);
*/
/*
  CU(hipMemcpy(p->v, v, n*n*sizeof(int_t), hipMemcpyDeviceToHost));
  CU(hipMemcpy(p->w, w, n*n*sizeof(int_t), hipMemcpyDeviceToHost));
  CU(hipMemcpy(p->wm, wm, n*n*sizeof(int_t), hipMemcpyDeviceToHost));
  CU(hipMemcpy(p->w5 - 1, w5 - 1, (n+1)*sizeof(int_t), hipMemcpyDeviceToHost));
  CU(hipMemcpy(p->w3, w3, (n+1)*sizeof(int_t), hipMemcpyDeviceToHost));
  frna_show(p);
*/

  CU(hipEventRecord(tstart,0));
//  for (d = 0; d < n; d++) { //for fragment lengths (1 : n) n-1 is the original way
  for (d = 0; d < n-1; d++) { //for fragment lengths (1 : n)

    #define ARGS d,n,dev_s,v,w,wm,w5,w3,dev_par
    #define TIMER_START CU(hipEventRecord(start,0));
    #define TIMER_STOP     CU(hipEventRecord(stop,0));\
                           CU(hipEventSynchronize(stop));\
                           CU(hipEventElapsedTime(&t,start,stop));
    TIMER_START
    calc_V_hairpin_and_V_stack<<<n,1>>>(ARGS);
    TIMER_STOP
    t1 += t;

    TIMER_START
    calc_V_bulge_internal<<<n,dim3(SQRT_NTHREAD,SQRT_NTHREAD,1)>>>(ARGS);
    TIMER_STOP
    t2 += t;

    TIMER_START
    calc_V_exterior<<<n,1>>>(ARGS);
//    hipError_t err = hipDeviceSynchronize();
  //  printf(hipGetErrorString(err);
    TIMER_STOP
    t3 += t;

    TIMER_START
    calc_V_multibranch<<<n,1>>>(ARGS);
    TIMER_STOP
    t4 += t;

#ifndef disablecoax
    TIMER_START
    calc_coaxial<<<n,NTHREAD>>>(ARGS);
    TIMER_STOP
    tcoax +=t;
#endif

    TIMER_START
    calc_W<<<n,1>>>(ARGS);
//    hipError_t err = hipDeviceSynchronize();
  //  printf("%s",hipGetErrorString(err));


    TIMER_STOP
    t5 += t;

    TIMER_START
    calc_WM<<<n,NTHREAD>>>(ARGS);
    TIMER_STOP
    t6 += t;
#ifndef disablecoax
    TIMER_START
    calc_wl_coax<<<n,NTHREAD>>>(ARGS ,wca);
    TIMER_STOP
    tcoax2 +=t;
#endif

    TIMER_START
    calc_w5_and_w3<<<1,NTHREAD>>>(ARGS,wca);
    TIMER_STOP
    t7 += t;

  }
  CU(hipEventRecord(tstop,0));
  CU(hipEventSynchronize(tstop));
  CU(hipEventElapsedTime(&t,tstart,tstop));

#ifdef TIMER
  fprintf(stderr, "Time for hairpin, stack: %3.1f ms\n", t1);
  fprintf(stderr, "Time for exterior: %3.1f ms\n", t2);
  fprintf(stderr, "Time for bulge/internal loop: %3.1f ms\n", t3);
  fprintf(stderr, "Time for multibranch: %3.1f ms\n", t4);
  fprintf(stderr, "Time for V coax: %3.1f ms\n", tcoax);
  fprintf(stderr, "Time for WM coax: %3.1f ms\n", tcoax2);
  fprintf(stderr, "Time for W: %3.1f ms\n", t5);
  fprintf(stderr, "Time for WM: %3.1f ms\n", t6);
  fprintf(stderr, "Time for w5 and w3: %3.1f ms\n", t7);
  fprintf(stderr, "Time for fill (total): %3.1f ms\n", t);

#endif
  CU(hipEventRecord(start,0));

  CU(hipMemcpy(p->v, v, n*n*sizeof(int_t), hipMemcpyDeviceToHost));
  CU(hipMemcpy(p->w, w, n*n*sizeof(int_t), hipMemcpyDeviceToHost));
  CU(hipMemcpy(p->wm, wm, n*n*sizeof(int_t), hipMemcpyDeviceToHost));
  CU(hipMemcpy(p->w5 - 1, w5 - 1, (n+1)*sizeof(int_t), hipMemcpyDeviceToHost));
  CU(hipMemcpy(p->w3, w3, (n+1)*sizeof(int_t), hipMemcpyDeviceToHost));

  CU(hipEventRecord(stop,0));
  CU(hipEventSynchronize(stop));
  CU(hipEventElapsedTime(&t,start,stop));

#ifdef TIMER
  fprintf(stderr, "Time for copy from GPU to CPU: %3.1f ms\n", t);
#endif

  CU(hipFree(v));
  CU(hipFree(w5 - 1));
  CU(hipFree(w3));
  CU(hipFree(w));
  CU(hipFree(wm));
  CU(hipEventDestroy(start));
  CU(hipEventDestroy(stop));
  CU(hipEventDestroy(tstart));
  CU(hipEventDestroy(tstop));

#else /* do serial fill on CPU */

#define ALLOC(a,sz) a = (int_t *) safe_malloc((sz)*sizeof(int_t))
#define ARGS d,p->n,p->seq,p->v,p->w,p->wm,p->w5,p->w3,par

/*  ALLOC(v,n*n); //best energy of structure closed by pair i,j. j>i: exterior fragment
  ALLOC(w,n*n); //best energy of structure from i to j
  ALLOC(wm,n*n); //best energy of structure i to j containing 2 or more branches
  ALLOC(w5,n+1); //best energy of structure from 1 to i
  w5++;//w5 is indexed from 1 -- is this a good idea?
  ALLOC(w3,n+1); //best energy of structure from i to numberofbases
*/


  init_w5_and_w3(n,p->w5,p->w3);
  init_w(n,p->w);

  int d;

  for (d = 0; d < n-1; d++) {

    calc_V_hairpin_and_V_stack(ARGS);

    calc_V_bulge_internal(ARGS);

    calc_V_exterior(ARGS);

    calc_V_multibranch(ARGS);
#ifndef disablecoax
    calc_coaxial(ARGS);
#endif
    calc_W(ARGS);

    calc_WM(ARGS);
#ifndef disablecoax
    calc_wl_coax(ARGS, p->wca);
#endif
    calc_w5_and_w3(ARGS, p->wca);

  }
printf("done with array fill\n");
  /*
  free(v);
  free(w);
  free(wm);
  free(w5-1);
  free(w3);
   */
#endif /* __HIPCC__ */
  return p;
} /* end frna_new */

void frna_delete(frna_t p)
{
  if (p) {
    if (p->seq)
      free(p->seq);
    if (p->v)
      free(p->v);
    if (p->w)
      free(p->w);
    if (p->wm)
      free(p->wm);
    if (p->w5 - 1)
      free(p->w5 - 1);
    if (p->w3)
      free(p->w3);
    free(p);
  }
}

#define SHOWARR(a)	 \
  if (p->a) {		      \
    int i, j;	      \
    for (i = 0; i < n; i++) { \
      printf("%s%4d: ",#a,i+1);				\
      for (j = 0; j < n; j++) {				\
	const int_t *aij = array_val(p->a,i,j,n,s);		\
	printf(RF" ", aij ? (*aij) : INF);	\
      }								\
      printf("\n");						\
    }								\
  }

#define SHOW(a)							\
  if (p->a) {								\
    int i;								\
    printf("%s: ",#a);							\
    for (i = 0; i < n; i++)						\
      printf(RF" ", p->a[i]);				\
    printf("\n");							\
  }									\
  //used to be mul by RT

/*void SHOW_2DARRAY(int_t* arr, int n)
{
  int i,j;
  printf("i\tj\tvalue\n");
  for(i=0;i<n;i++)
    for(j=0;j<n;j++)
      printf("%d\t%d\t%d\n",i,j,arr[ind(i,j,n)]);
}
*/

void frna_show(const frna_t p)
{
  int i,j, n = p->n;
  const fbase_t *s = p->seq;
  printf("n: %d\n", n);
  printf("seq: ");
  for (i = 0; i < n; i++)
    printf("%c", fbase_as_char(s[i]));
  printf("\n");
  printf("i\tj\tV:\tW:\tWM:\tV':\tW':\tWM':\n");
  for(j=0;j<n;j++)
    for(i=0;i<j;i++)
        printf("%d\t%d\t%d\t%d\t%d\t%d\t%d\t%d\n",i+1,j+1,p->v[ind(i,j,n)],p->w[ind(i,j,n)],p->wm[ind(i,j,n)],p->v[ind(j,i,n)],p->w[ind(j,i,n)],p->wm[ind(j,i,n)] );
        //printf("%d\t%d\t%d\n",i+1,j+1,p->w[ind(j,i,n)]);


/*
SHOWARR(w);
  SHOWARR(wm);
*/
//  SHOW(w5);
//  SHOW(w3);
  printf("\n\n\ni\tw5[i]\tw3[i]\n");
  printf("0\t0\t0\n");
  for(i=0;i<n;i++){
    printf("%d\t",i+1);
    printf("%d\t",p->w5[i]);
    printf("%d\n",p->w3[i]);
  }

}

short base_as_num(fbase_t b)
{
  switch (b) {
  case A:
    return 1;
  case C:
    return 2;
  case G:
    return 3;
  case U:
    return 4;
  default:
    printf("unknown base %d\n",b);
    die("base_as_num: unknown base");
    return 0;
  }

}

fbase_t num_as_base(short x)
{
  switch (x) {
  case 1:
    return A;
  case 2:
    return C;
  case 3:
    return G;
  case 4:
    return U;
  default:
//    die("base_as_num: unknown base");
    return A;
  }
}
#ifdef SHORT
int int_pow(int val,int exp)//val^exp, integer type
{
  if (exp<0) die("int_t_pow:tried to take negative exponent");
  if (exp==0) return 1;
  else if (exp==1) return val;
  return val * int_pow(val,exp-1);
}
#endif
int_t int_t_pow(int_t val,int_t exp)//val^exp, integer type
{
  if (exp<0) die("int_t_pow:tried to take negative exponent");
  if (exp==0) return 1;
  else if (exp==1) return val;
  return val * int_t_pow(val,exp-1);
}

#define write_int_t(x) fwrite(x,sizeof(int_t),1,savefile)
#define write_int(x) fwrite(x,sizeof(int),1,savefile)
#define write_short(x) fwrite(x,sizeof(short),1,savefile)
#define write_char(x) fwrite(x,sizeof(char),1,savefile)
#define write_float(x) fwrite(x,sizeof(float),1,savefile)

#define b(x) num_as_base(x)

void frna_write_save_file(const frna_t p, const fparam_t par, const char* outfile)
{
  int i,j,k,l,m,n,o,q;
  FILE *savefile;
  int_t zero=0;
  int intzero = 0;
  int_t infinity=INF;
  char FALSE=0;
  int inc[6][6]={{0,0,0,0,0,0},{0,0,0,0,1,0},{0,0,0,1,0,0},{0,0,1,0,1,0},
                                                {0,1,0,1,0,0},{0,0,0,0,0,0}};//"can-pair" array
  savefile = fopen(outfile,"wb");
  if (!savefile) die("failed to open output file");
//save file version
  short vers=4;//this is safiversion from defines.h
  fwrite(&vers,sizeof(short),1,savefile);

//basic structure information
  int sequencelength = p->n;
  char intermolecular=0; //fold-cuda does not currentlty suport intermolecular folding
  fwrite(&sequencelength,sizeof(int),1,savefile);
  fwrite(&intermolecular,sizeof(char),1,savefile);

  int pairs = 0; //we do not currently output any pairs, just the energies for dotplot creation
  fwrite(&pairs,sizeof(int),1,savefile);
  int forbiddenpairs=0;//likewise
  fwrite(&forbiddenpairs,sizeof(int),1,savefile);

//write the nucleotide sequence. hnumber is the base position
  char nuc=0;
  short hnum;
  for(i=0;i<=p->n;i++){
    if(i>0) nuc = fbase_as_char(p->seq[i-1]);//i-1 because seq is 0-indexed but rnastructure expects 1-indexed
    hnum = i;
  	fwrite(&hnum,sizeof(short),1,savefile);//hnumber[0] is nothing, as is nuc[0]
    fwrite(&nuc,sizeof(char),1,savefile);
  }

  short numseq[2*p->n+1];//numseq array contains the sequnece in a different format
  for(i=1;i<=p->n;i++){
    numseq[i] = numseq[i+p->n] = base_as_num(p->seq[i-1])-1;
  }
  for(i=0;i<=2*p->n;i++)
    fwrite(&numseq[i],sizeof(short),1,savefile);

  int doubles=0;//forced double stranded nucs, not supported here
  fwrite(&doubles, sizeof(int),1,savefile);

  int singles=0;//forced single stranded nucleotides, we don't have those either
  fwrite(&singles,sizeof(int),1,savefile);

  int modified=0;//or modifed based
  fwrite(&modified,sizeof(int),1,savefile);

  int numberGU=0;//or nucs constrained to be in GU pairs
  fwrite(&numberGU,sizeof(int),1,savefile);

  const char* label="sequence";
  int length = (int) strlen(label);
  //write_int(&length);
  write_int(&intzero);
  //fwrite(label,sizeof(char),length,savefile);

  char templated=0;
  fwrite(&templated,sizeof(char),1,savefile);//we're not constrained according to a template

  char shaped=0;
  fwrite(&shaped,sizeof(char),1,savefile);//we're not constrained by SHAPE data

//int testnum=55;//for debugging
//fwrite(&testnum,sizeof(int),1,savefile);
  int jtmp;
//now, write the arrays
  int_t w5=0,w3=0,v=0,w=0,wm=0;
  char fce=0;
  for(i=0;i<=p->n;i++){
    if (i>0) {w5=p->w5[i-1]; w3 = p->w3[i-1];}
  		write_int_t(&w3);
  		write_int_t(&w5);
    for(jtmp=0;jtmp<=p->n;jtmp++){
      j= jtmp+i > p->n? jtmp+i-p->n:jtmp+i;
      if((i>0) && (jtmp>0)){
        v = p->v[ind(i-1,j-1,p->n)];
        w = p->w[ind(i-1,j-1,p->n)];
        wm = p->wm[ind(i-1,j-1,p->n)];
      }
      else v=w=wm=14000;
  		write_int_t(&v);
  		write_int_t(&w);
  		write_int_t(&wm);
  		write_char(&fce);

    }
  }


  write_int_t(&p->w3[p->n]);
//int_t six=6;
//int_t seven=7;
  for(i=0;i<=2*p->n;i++){
  	write_char(&FALSE);//fce[i]
    write_char(&FALSE);//mod[i]
  }

//write vmin
int_t vmintmp = INF;
//  int vmin = (int) p->w3[0];
  for(i=0;i<p->n-1;i++){
 //   printf("\ni=%d ",i);
    for(j=i+1;j<p->n;j++) {
   //   printf( "j=%d ",j);
      vmintmp = int_t_min(vmintmp, p->v[ind(i,j,p->n)]+p->v[ind(j,i,p->n)]);
    }
  }
int vmin = (int) vmintmp;
//printf("vmin=%d\n",vmin);

write_int(&vmin);

//now, write a the thermodynamic data in a format that refold wants

  // First write the alphabet information, equivalent to the following lines from rna_library.cpp 
  // write(out, &(data->alphabet));
	
  // data->alphabet is a vector of vectors

  i = 4; //alphabet size
	write_int(&i);
  
  i=2; // number of characters for A, C, G, and U
  // Write out vector of characters for A
  write_int(&i);
  nuc = 'a';
  write_char(&nuc);
  nuc = 'A';
  write_char(&nuc);

  // Write out vector of characters for C
  write_int(&i);
  nuc = 'c';
  write_char(&nuc);
  nuc = 'C';
  write_char(&nuc);

  // Write out vector of characters for G
  write_int(&i);
  nuc = 'g';
  write_char(&nuc);
  nuc = 'G';
  write_char(&nuc);

  // Write out vector of characters for U
  write_int(&i);
  nuc = 'u';
  write_char(&nuc);
  nuc = 'U';
  write_char(&nuc);

  // End writing alphabet

  // write(out, &(data->pairing));
  bool paired=true;
  bool unpaired=false;
  int num_bases = 4;
  int num_dangles = 3;

  // data-> pairing is a vector of vectors
  write_int(&num_bases);
  for (i=1; i<=num_bases;i++){
    write_int(&num_bases);
    for (j=1; j<=num_bases;j++){
      if (is_canonical_pair(b(i), b(j)))
        fwrite(&paired,sizeof(bool),1,savefile);
      else
        fwrite(&unpaired,sizeof(bool),1,savefile);
    }
  }

  // write_int(&num_bases);
  
  // write_int(&num_bases);
  // fwrite(&unpaired,sizeof(bool),1,savefile);
  // fwrite(&unpaired,sizeof(bool),1,savefile);
  // fwrite(&unpaired,sizeof(bool),1,savefile);
  // fwrite(&paired,sizeof(bool),1,savefile);

  // write_int(&num_bases);
  // fwrite(&unpaired,sizeof(bool),1,savefile);
  // fwrite(&unpaired,sizeof(bool),1,savefile);
  // fwrite(&paired,sizeof(bool),1,savefile);
  // fwrite(&unpaired,sizeof(bool),1,savefile);

  // write_int(&num_bases);
  // fwrite(&unpaired,sizeof(bool),1,savefile);
  // fwrite(&paired,sizeof(bool),1,savefile);
  // fwrite(&unpaired,sizeof(bool),1,savefile);
  // fwrite(&paired,sizeof(bool),1,savefile);

  // write_int(&num_bases);
  // fwrite(&paired,sizeof(bool),1,savefile);
  // fwrite(&unpaired,sizeof(bool),1,savefile);
  // fwrite(&paired,sizeof(bool),1,savefile);
  // fwrite(&unpaired,sizeof(bool),1,savefile);

  // data->not_pairing, data->non_interacting, and data->linker are for features not supported by Fold-cuda
  // instead write out equivalent of zero-sized vectors

  // write(out, &(data->not_pairing));
  i = 0;
	write_int(&i);
	// write(out, &(data->non_interacting));
  i = 0;
	write_int(&i);
	// write(out, &(data->linker));
  i = 0;
	write_int(&i);

  int_t poppen[5] = {0,6,6,6,6};
  i = 5;
	write_int(&i); //write size of vector
  for (i=0;i<5;i++) write_int_t(&poppen[i]); //data->poppen[i] //used in partition function

  write_int_t(&par->maximum_correction); //data->maxpen

  int_t eparam[11] = {0,0,0,0,0,par->a,par->b,30,30,-500,par->c};
  i = 11;
	write_int(&i); //write size of vector
  for (i=0;i<11;i++) write_int_t(&eparam[i]); //data->eparam[i]
  
  i = 31;
	write_int(&i); //write size of vector
  for (i=0;i<31;i++) {
    write_int_t(&par->internal_loop_initiation[i]);//data->inter[i]
  }

	write_int(&i); //write size of vector
  for (i=0;i<31;i++) {
    write_int_t(&par->bulge_loop_initiation[i]);//data->bulge[i]
  }

	write_int(&i); //write size of vector
  for (i=0;i<31;i++) {
    write_int_t(&par->hairpin_loop_initiation[i]);//data->hairpin[i]
  }

  // Write out dangle
  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_dangles); //write size of vector
        write_int_t(&zero);//data->dangle[i][j][k][l] l[0] is meaningless
        write_int_t(&par->dangle_3p[num_as_base(i)][num_as_base(j)][num_as_base(k)]);
        write_int_t(&par->dangle_5p[num_as_base(i)][num_as_base(j)][num_as_base(k)]);
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          // if(i&&j&&k&&l)write_int_t(&par->stack[b(i)][b(j)][b(k)][b(l)]);//data->stack[i][j][k][l]
          // else write_int_t(&zero);
          write_int_t(&par->stack[b(i)][b(j)][b(k)][b(l)]);//data->stack[i][j][k][l]
        }
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          // if(i&&j&&k&&l)write_int_t(&par->tstackh[b(i)][b(j)][b(k)][b(l)]);//data->tstckh[i][j][k][l]
          // else write_int_t(&zero);
          write_int_t(&par->tstackh[b(i)][b(j)][b(k)][b(l)]);//data->tstackh[i][j][k][l]
        }
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          // if(i&&j&&k&&l)write_int_t(&par->tstacki[b(i)][b(j)][b(k)][b(l)]);//data->tstcki[i][j][k][l]
          // else write_int_t(&zero);
          write_int_t(&par->tstacki[b(i)][b(j)][b(k)][b(l)]);//data->tstcki[i][j][k][l]
        }
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          // if(i&&j&&k&&l)write_int_t(&par->coaxial[b(i)][b(j)][b(k)][b(l)]);//data->coax[i][j][k][l]
          // else write_int_t(&zero);
          write_int_t(&par->coaxial[b(j)][b(i)][b(k)][b(l)]);//data->coaxial[i][j][k][l] i and j indices are switched for coaxial (see read_stack function in fparam.c)
        }
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          // if(i&&j&&k&&l)write_int_t(&par->tstackcoax[b(i)][b(j)][b(k)][b(l)]);
          // else write_int_t(&zero);
          write_int_t(&par->tstackcoax[b(i)][b(j)][b(k)][b(l)]);//data->tstackcoax[i][j][k][l]
        }
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          // if(i&&j&&k&&l)write_int_t(&par->coaxstack[b(i)][b(j)][b(k)][b(l)]);
          // else write_int_t(&zero);
          write_int_t(&par->coaxstack[b(i)][b(j)][b(k)][b(l)]);//data->coaxstack[i][j][k][l]
        }
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          // if(i&&j&&k&&l)write_int_t(&par->tstack[b(i)][b(j)][b(k)][b(l)]);
          // else write_int_t(&zero);
          write_int_t(&par->tstack[b(i)][b(j)][b(k)][b(l)]);//data->tstack[i][j][k][l]
        }
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          // if(i&&j&&k&&l)write_int_t(&par->tstackm[b(i)][b(j)][b(k)][b(l)]);
          // else write_int_t(&zero);
          write_int_t(&par->tstackm[b(i)][b(j)][b(k)][b(l)]);//data->tstackm[i][j][k][l]
        }
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          // if(i&&j&&k&&l)write_int_t(&par->tstacki23[b(i)][b(j)][b(k)][b(l)]);
          // else write_int_t(&zero);
          write_int_t(&par->tstacki23[b(i)][b(j)][b(k)][b(l)]);//data->tstacki23[i][j][k][l]
        }
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          // if(i&&j&&k&&l)write_int_t(&par->tstacki1n[b(i)][b(j)][b(k)][b(l)]);
          // else write_int_t(&zero);
          write_int_t(&par->tstacki1n[b(i)][b(j)][b(k)][b(l)]);//data->tstacki1n[i][j][k][l]
        }
      }
    }
  }

  write_int(&num_bases); //write size of vector
  for (i=1;i<=num_bases;i++) {
    write_int(&num_bases); //write size of vector
    for (j=1;j<=num_bases;j++) {
      write_int(&num_bases); //write size of vector
      for (k=1;k<=num_bases;k++) {
        write_int(&num_bases); //write size of vector
        for (l=1;l<=num_bases;l++) {
          write_int(&num_bases); //write size of vector
          for (m=1;m<=num_bases;m++) {
            write_int(&num_bases); //write size of vector
            for (n=1;n<=num_bases;n++) {
              // if(i&&j&&k&&l&&m&&n)write_int_t(&par->int11[b(i)][b(j)][b(k)][b(n)][b(m)][b(l)]);//iloop11
              // else write_int_t(&infinity);
              write_int_t(&par->int11[b(i)][b(j)][b(k)][b(n)][b(m)][b(l)]);//data->stack[i][j][k][l]
            }
          }
        }
      }
    }
  }


  // int21 (iloop21) and int22 (iloop22) data tables are not vectorized
  for (i=1;i<=num_bases;i++) {
    for (j=1;j<=num_bases;j++) {
      for (k=1;k<=num_bases;k++) {
        for (l=1;l<=num_bases;l++) {
          for (m=1;m<=num_bases;m++) {
            for (n=1;n<=num_bases;n++) {
              for (o=1;o<=num_bases;o++) {
                if (inc[i][j]&&inc[n][o]){
                  // if(i&&j&&k&&l&&m&&n&&o)write_int_t(&par->int21[b(i)][b(j)][b(k)][b(l)][b(m)][b(n)][b(o)]);//iloop21
                  // else write_int_t(&infinity);
                  if (is_canonical_pair(b(i), b(j)) && is_canonical_pair(b(n), b(o)))
                    write_int_t(&par->int21[b(i)][b(j)][b(k)][b(l)][b(m)][b(n)][b(o)]);//iloop21
                  
                }
                for (q=1;q<=num_bases;q++) {
                  if (inc[i][k]&&inc[j][l]){
                    // if(i&&j&&k&&l&&m&&n&&o&&q&&(i!=5)&&(i!=5)&&(j!=5)&&(k!=5)&&(l!=5)&&(m!=5)&&(n!=5)&&(o!=5)&&(q!=5))
                    //   write_int_t(&par->int22[b(i)][b(j)][b(k)][b(l)][b(m)][b(n)][b(o)][b(q)]);//iloop222
                    // else write_int_t(&infinity);
                    if (is_canonical_pair(b(i), b(k)) && is_canonical_pair(b(j), b(l)))
                      write_int_t(&par->int22[b(i)][b(j)][b(k)][b(l)][b(m)][b(n)][b(o)][b(q)]);//iloop222
                  }
                }
              }
            }
          }
        }
      }
    }
  }

//translate tetrsaloop to rnastructure format
  int_t ntloop = (int_t) par->ntloop;
  int_t tloop[par->ntloop+1][2];
  for(i=0;i<=par->ntloop;i++){
    tloop[i][0]=tloop[i][1]=0;
    for(j=0;j<6;j++){//6nt in a tetraloop
      if (i!=0) tloop[i][0] += (base_as_num(par->tloop[i-1].seq[j]) * int_t_pow(num_bases,j));
    }
    if (i!=0) tloop[i][1] = par->tloop[i-1].val;
    else tloop[i][1]=0;
  }
//write tetraloop
  short num_loops = (short) par->ntloop;
  write_short(&num_loops);//data->numoftloops));
  i = (int) ntloop+1;
  k=2;
  write_int(&i);//data->numoftloops));
  for (i=0;i<=par->ntloop;i++) {
    write_int(&k);//length of a tetraloop
    j = (int) tloop[i][0];
    write_int(&j);
    j = (int) tloop[i][1];
    write_int(&j);
    
    // for (j=0;j<k;j++) write_int_t(&tloop[i][j]);
  }

//translate triloop to rnastructure format
  int_t ntriloop = (int_t) par->ntriloop;//we have them as ints, they need to be shorts
  int_t triloop[par->ntriloop+1][2];
  for(i=0;i<=par->ntriloop;i++){
    triloop[i][0]=triloop[i][1]=0;
    for(j=0;j<5;j++){//5nt in a triloop
      if (i!=0) triloop[i][0] += (base_as_num(par->triloop[i-1].seq[j]) * int_t_pow(num_bases,j));
    }
    if (i!=0) triloop[i][1] = par->triloop[i-1].val;
  }
//write triloop data
  num_loops = (short) par->ntriloop;
  write_short(&num_loops);//data->numoftloops));
  i = (int) num_loops+1;
  write_int(&i);//data->numoftriloops));
  for (i=0;i<=par->ntriloop;i++) {
    write_int(&k);
    j = (int) triloop[i][0];
    write_int(&j);
    j = (int) triloop[i][1];
    write_int(&j);
    // for (j=0;j<k;j++) write_int_t(&triloop[i][j]);
  }

//translate hexaloop to rnastructure format
  int_t nhexaloop = (int_t) par->nhexaloop;//we have them as ints, they need to be shorts
  int hexaloop[par->nhexaloop+1][2];//has to be an int because the numbers to represent sequence get big
  for(i=0;i<=par->nhexaloop;i++){
    hexaloop[i][0]=triloop[i][1]=0;
    for(j=0;j<8;j++){//5nt in a triloop
#ifdef SHORT//if int_t is int, then we use the int_t_pow function instead of int_pow (there's probably a better way to do this)
      if (i!=0) hexaloop[i][0] += (base_as_num(par->hexaloop[i-1].seq[j]) * int_pow(num_bases,j));
#else
      if (i!=0) hexaloop[i][0] += (int) (base_as_num(par->hexaloop[i-1].seq[j]) * int_t_pow(num_bases,j));
#endif
    }
    if (i!=0) hexaloop[i][1] = (int) par->hexaloop[i-1].val;
  }
//write hexaloop data
  num_loops = (short) par->nhexaloop;
  write_short(&num_loops);//data->numoftloops));
  i = (int) num_loops+1;
  write_int(&i);//data->numofhexaloops));
  for (i=0;i<=par->nhexaloop;i++) {
    write_int(&k);
    j = (int) hexaloop[i][0];
    write_int(&j);
    j = (int) hexaloop[i][1];
    write_int(&j);
    // for (j=0;j<k;j++) write_int(&hexaloop[i][j]);
  }

  write_int_t(&par->terminal_AU_penalty); //data->auend
  write_int_t(&par->bonus_for_GGG_hairpin); //data->gubonus
  write_int_t(&par->c_hairpin_intercept); //data->cint
  write_int_t(&par->c_hairpin_slope); //data->cslope
  write_int_t(&par->c_hairpin_of_3); //data->c3
  write_int_t(&par->a);//data->efn2a
  write_int_t(&par->b);//data->efn2b
  write_int_t(&par->c);//data->efn2c
  write_int_t(&zero); //data->init (we skip)
  write_int_t(&zero); //mlasym (we skip)
  write_int_t(&zero); //strain (we skip)
//  float prelog = (float) par->Extrapolation_for_large_loops;//rnastructure expects this as a float
  write_float(&par->prelog); //data->prelog
  write_int_t(&par->Bonus_for_Single_C_bulges_adjacent_to_C);//data->singlecbulge

  fclose(savefile);
}

